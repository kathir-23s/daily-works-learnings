#include <iostream>
#include <hip/hip_runtime.h>

__global__ void add_kernel(const float* A, const float* B, float* C, int N){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N){
        C[idx] = A[idx] + B[idx];
    }
}

extern "C" {
    void add_matrix_gpu(const float* A, const float* B, float* C, int N){
        float *d_A, *d_B, *d_C;

        hipMalloc((void**)&d_A, N * sizeof(float));
        hipMalloc((void**)&d_B, N * sizeof(float));
        hipMalloc((void**)&d_C, N * sizeof(float));

        hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

        dim3 blockSize(160);
        dim3 gridSize((N + blockSize.x - 1) / blockSize.x);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        add_kernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float ms = 0;
        hipEventElapsedTime(&ms, start, stop);

        hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

        std::cout << "[GPU] Matrix addition took " << ms << " ms\n";

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
}