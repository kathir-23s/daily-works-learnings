#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void wxb_kernel(const float* W, const float* X, const float* b, float* Y,
                           int out_features, int in_features, int batch_size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  
    int col = blockIdx.x * blockDim.x + threadIdx.x;  

    if (row < batch_size && col < out_features) {
        float sum = 0.0f;
        for (int k = 0; k < in_features; k++) {
            sum += X[row * in_features + k] * W[col * in_features + k];
        }
        Y[row * out_features + col] = sum + b[col];
    }
}

extern "C" void wxb_gpu(const float* W, const float* X, const float* b, float* Y,
                        int out_features, int in_features, int batch_size, float* time_ms) {
    float *d_W, *d_X, *d_b, *d_Y;
    size_t sizeW = out_features * in_features * sizeof(float);
    size_t sizeX = batch_size * in_features * sizeof(float);
    size_t sizeB = out_features * sizeof(float);
    size_t sizeY = batch_size * out_features * sizeof(float);

    hipMalloc(&d_W, sizeW);
    hipMalloc(&d_X, sizeX);
    hipMalloc(&d_b, sizeB);
    hipMalloc(&d_Y, sizeY);

    hipMemcpy(d_W, W, sizeW, hipMemcpyHostToDevice);
    hipMemcpy(d_X, X, sizeX, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeB, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((out_features + 15) / 16, (batch_size + 15) / 16);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    wxb_kernel<<<blocks, threads>>>(d_W, d_X, d_b, d_Y, out_features, in_features, batch_size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(time_ms, start, stop);

    hipMemcpy(Y, d_Y, sizeY, hipMemcpyDeviceToHost);

    hipFree(d_W);
    hipFree(d_X);
    hipFree(d_b);
    hipFree(d_Y);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}
