// mul_gpu.cu
#include <iostream>
#include <hip/hip_runtime.h>

extern "C" {

__global__ void matmul_kernel(const float* A, const float* B, float* C, int rowsA, int colsA, int colsB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rowsA && col < colsB) {
        float sum = 0.0f;
        for (int k = 0; k < colsA; k++) {
            sum += A[row * colsA + k] * B[k * colsB + col];
        }
        C[row * colsB + col] = sum;
    }
}

void matmul_gpu(const float* A, const float* B, float* C, int rowsA, int colsA, int colsB, float* time_ms) {
    float *d_A, *d_B, *d_C;
    size_t sizeA = rowsA * colsA * sizeof(float);
    size_t sizeB = colsA * colsB * sizeof(float);
    size_t sizeC = rowsA * colsB * sizeof(float);

    hipMalloc(&d_A, sizeA);
    hipMalloc(&d_B, sizeB);
    hipMalloc(&d_C, sizeC);

    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((colsB + 15) / 16, (rowsA + 15) / 16);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matmul_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rowsA, colsA, colsB);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(time_ms, start, stop);

    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

}
