#include <hip/hip_runtime.h>
#include <iostream>

extern "C" {

__global__ void wxb_kernel(const float* W, const float* X, const float* b, float* Y,
                           int out_features, int in_features, int batch_size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // batch index
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // output feature index

    if (row < batch_size && col < out_features) {
        float sum = 0.0f;
        for (int k = 0; k < in_features; k++) {
            sum += X[row * in_features + k] * W[col * in_features + k];
        }
        Y[row * out_features + col] = sum + b[col];
    }
}

void wxb_gpu(const float* W, const float* X, const float* b, float* Y,
             int out_features, int in_features, int batch_size, float* time_ms) {
    float *d_W, *d_X, *d_b, *d_Y;
    size_t sizeW = out_features * in_features * sizeof(float);
    size_t sizeX = batch_size * in_features * sizeof(float);
    size_t sizeB = out_features * sizeof(float);
    size_t sizeY = batch_size * out_features * sizeof(float);

    hipMalloc(&d_W, sizeW);
    hipMalloc(&d_X, sizeX);
    hipMalloc(&d_b, sizeB);
    hipMalloc(&d_Y, sizeY);

    hipMemcpy(d_W, W, sizeW, hipMemcpyHostToDevice);
    hipMemcpy(d_X, X, sizeX, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeB, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((out_features + 15) / 16, (batch_size + 15) / 16);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    wxb_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_W, d_X, d_b, d_Y,
                                                   out_features, in_features, batch_size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(time_ms, start, stop);
    hipMemcpy(Y, d_Y, sizeY, hipMemcpyDeviceToHost);

    hipFree(d_W);
    hipFree(d_X);
    hipFree(d_b);
    hipFree(d_Y);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

}
